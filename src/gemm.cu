#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand.h>
#include <hip/driver_types.h>
#include <time.h>
#include <cstdlib>
#include <iostream>
#include <ostream>

//fill gpu with random numbers
void GPU_fill_rand(float *A, int n_rows_A, int n_cols_A) {

	//pseudo-random number generator
	hiprandGenerator_t prng;
	hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);

	//set seed
	hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());

	//fill the array with random numbers on the gpu
	hiprandGenerateUniform(prng, A, n_rows_A * n_cols_A);
}

//On the gpu multiply C(m,n)=A(m,k)*B(k,n)
void gpu_blas_mmul(const float *A, const float *B, float *C, const int m,
		const int k, const int n) {

	int lda = m, ldb = k, ldc = m;
	const float alf = 1;
	const float bet = 0;
	const float *alpha = &alf;
	const float *beta = &bet;

	//cublas handle
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	//do the actual multiplication
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B,
			ldb, beta, C, ldc);

	//destroy the handle
	hipblasDestroy(handle);

}

//print matrix
void print_matrix(const float *A, int nr_rows_A, int nr_cols_A) {

	for (int i = 0; i < nr_rows_A; ++i) {
		for (int j = 0; j < nr_cols_A; ++j) {
			std::cout << A[j * nr_rows_A + i] << " ";
		}
		std::cout << std::endl;
	}
	std::cout << std::endl;
}

int main() {

	//Allocate 3 arrays on CPU
	int n_rows_A, n_cols_A, n_rows_B, n_cols_B, n_rows_C, n_cols_C;

	//Try square matrices
	n_rows_A = n_cols_A = n_rows_B = n_cols_B = n_rows_C = n_cols_C = 3;

	//actually allocate on CPU
	float *h_A = (float *) malloc(n_rows_A * n_cols_A * sizeof(float));
	float *h_B = (float *) malloc(n_rows_B * n_cols_B * sizeof(float));
	float *h_C = (float *) malloc(n_rows_C * n_cols_C * sizeof(float));

	//allocate 3 arrays on GPU
	float *d_A, *d_B, *d_C;
	hipMalloc(&d_A, n_rows_A * n_cols_A * sizeof(float));
	hipMalloc(&d_B, n_rows_B * n_cols_B * sizeof(float));
	hipMalloc(&d_C, n_rows_C * n_cols_C * sizeof(float));

	//actually fill the allocated arrays
	GPU_fill_rand(d_A, n_rows_A, n_cols_A);
	GPU_fill_rand(d_B, n_rows_B, n_cols_B);

	//copy the filled arrays back to the cpu to print them
	hipMemcpy(h_A, d_A, n_rows_A * n_cols_A * sizeof(float),
			hipMemcpyDeviceToHost);
	hipMemcpy(h_B, d_B, n_rows_B * n_cols_B * sizeof(float),
			hipMemcpyDeviceToHost);

	//print A and B matrices
	std::cout << "A = " << std::endl;
	print_matrix(h_A, n_rows_A, n_cols_A);
	std::cout << "B = " << std::endl;
	print_matrix(h_A, n_rows_A, n_cols_A);

	//multiply on gpu
	gpu_blas_mmul(d_A, d_B, d_C, n_rows_A, n_cols_A, n_cols_B);

	//copy multiplication result back to cpu
	hipMemcpy(h_C,d_C,n_rows_C * n_cols_C * sizeof(float),hipMemcpyDeviceToHost);

	std::cout << "C = " << std::endl;
	print_matrix(h_C, n_rows_C, n_cols_C);

	//free gpu memory
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	//free cpu memory
	free(h_A);
	free(h_B);
	free(h_C);

	return 0;
}
